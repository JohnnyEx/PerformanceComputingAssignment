#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#include "vtk.h"
#include "data.h"
#include "setup.h"

/**
 * @brief Set up some default values before arguments have been loaded
 * 
 */
void set_defaults() {
	m_variables.lengthX = 1.0;
	m_variables.lengthY = 1.0;

	m_variables.X = 4000;
	m_variables.Y = 4000;

    graph.grid_x = 10;
    graph.grid_y = 10;
    graph.block_x = 50;
    graph.block_y = 50;

	T = 1.6e-9;

	set_default_base();
}

/**
 * @brief Set up some of the values required for computation after arguments have been loaded
 * 
 */
void setup() {
	m_variables.dx = m_variables.lengthX / m_variables.X;
	m_variables.dy = m_variables.lengthY / m_variables.Y;

	m_variables.dt = m_constants.cfl * (m_variables.dx > m_variables.dy ? m_variables.dx : m_variables.dy) / m_constants.c;
	
	if (steps == 0) // only set this if steps hasn't been specified
		steps = (int) (T / m_variables.dt);
}

/**
 * @brief Allocate all of the arrays used for computation
 * 
 */
void allocate_arrays() {
    m_arrays.Ex_size_x = m_variables.X; m_arrays.Ex_size_y = m_variables.Y+1;
    alloc_2d_array(m_variables.X, m_variables.Y+1, &m_arrays.Ex, &m_arrays.ex_pitch);
    m_arrays.Ey_size_x = m_variables.X+1; m_arrays.Ey_size_y = m_variables.Y;
    alloc_2d_array(m_variables.X+1, m_variables.Y, &m_arrays.Ey, &m_arrays.ey_pitch);
    m_arrays.Bz_size_x = m_variables.X; m_arrays.Bz_size_y = m_variables.Y;
    alloc_2d_array(m_variables.X, m_variables.Y, &m_arrays.Bz, &m_arrays.bz_pitch);

    m_arrays.E_size_x = m_variables.X+1; m_arrays.E_size_y = m_variables.Y+1; m_arrays.E_size_z = 3;
    alloc_3d_cuda_array(m_arrays.E_size_x, m_arrays.E_size_y, m_arrays.E_size_z, &m_arrays.E, &m_arrays.e_pitch);
    host_E = alloc_3d_array(m_arrays.E_size_x, m_arrays.E_size_y, m_arrays.E_size_z);
    m_arrays.B_size_x = m_variables.X+1; m_arrays.B_size_y = m_variables.Y+1; m_arrays.B_size_z = 3;
    alloc_3d_cuda_array(m_arrays.B_size_x, m_arrays.B_size_y, m_arrays.B_size_z, &m_arrays.B, &m_arrays.b_pitch);
    host_B = alloc_3d_array(m_arrays.B_size_x, m_arrays.B_size_y, m_arrays.B_size_z);
}

/**
 * @brief Free all of the arrays used for the computation
 * 
 */
void free_arrays() {
	free_2d_array(m_arrays.Ex);
	free_2d_array(m_arrays.Ey);
	free_2d_array(m_arrays.Bz);
    free_3d_cuda_array(m_arrays.E);
    free_3d_cuda_array(m_arrays.B);
	free_3d_array(host_E);
	free_3d_array(host_B);
}

/**
 * @brief Set up a guassian to curve around the centre
 * 
 */
__global__ void problem_set_up(variables m_variables, arrays m_arrays) {
    double xcen = m_variables.lengthX / 2.0;
    double ycen = m_variables.lengthY / 2.0;

    for (int i = 0; i < m_arrays.Ex_size_x; i++ ) {
        for (int j = 0; j < m_arrays.Ex_size_y; j++) {
            double xcoord = (i - xcen) * m_variables.dx;
            double ycoord = j * m_variables.dy;
            double rx = xcen - xcoord;
            double ry = ycen - ycoord;
            double rlen = sqrt(rx*rx + ry*ry);
            double tx = (rlen == 0) ? 0 : ry / rlen;
            double mag = exp(-400.0 * (rlen - (m_variables.lengthX / 4.0)) * (rlen - (m_variables.lengthY / 4.0)));
            m_arrays.Ex[i * m_arrays.ex_pitch + j] = mag * tx;
        }
    }
    for (int i = 0; i < m_arrays.Ey_size_x; i++ ) {
        for (int j = 0; j < m_arrays.Ey_size_y; j++) {
            double xcoord = i * m_variables.dx;
            double ycoord = (j - ycen) * m_variables.dy;
            double rx = xcen - xcoord;
            double ry = ycen - ycoord;
            double rlen = sqrt(rx*rx + ry*ry);
            double ty = (rlen == 0) ? 0 : -rx / rlen;
            double mag = exp(-400.0 * (rlen - (m_variables.lengthY / 4.0)) * (rlen - (m_variables.lengthY / 4.0)));
            m_arrays.Ey[i*m_arrays.ey_pitch + j] = mag * ty;
        }
    }
}
