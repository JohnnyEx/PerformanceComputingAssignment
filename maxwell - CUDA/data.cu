#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>

#include "vtk.h"
#include "data.h"

struct constants m_constants = {
        .c = 299792458, // Speed of light
        .mu = 4.0 * M_PI * 1.0e-7, // permiability of free space
        .eps = 1.0 / (m_constants.c * m_constants.c * m_constants.mu), // permitivitty of free space
        .cfl = 0.6363961031,

};

struct variables m_variables;
struct arrays m_arrays;
struct cudaGraph graph;

double *** host_E;
double *** host_B;

// Time to run for / or number of steps
double T = 0.0001;
int steps = 0;

/**
 * @brief Allocate a 2D array that is addressable using square brackets - now this is a Cuda one
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @param array The array that will be populated and will be allocated the space necessary to do so
 * @param pitch The pitch value that will be calculated of the array given in case
 */
void alloc_2d_array(int m, int n, double **array, size_t *pitch) {
  	hipMallocPitch((void **)array, pitch, n * sizeof(double), m);
    *pitch = (*pitch) / sizeof (double);
}

/**
 * @brief Free a 2D array - now cuda
 * 
 * @param array The 2D array to free
 */
void free_2d_array(double * array) {
    hipFree(array);
}

/**
 * @brief Allocate a 3D array that is addressable using square brackets
 * 
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @param o The third dimension of the array
 * @return double*** A 3D array
 */
double ***alloc_3d_array(int m, int n, int o) {
	double ***x;
	x = (double***) malloc(m*sizeof(double **));
	x[0] = (double **) malloc(m*n*sizeof(double *));
	x[0][0] = (double *) calloc(m*n*o,sizeof(double));
	for (int i = 1; i < m; i++) {
		x[i] = &x[0][i*n];
	}
	for (int i = 0; i < m; i++) {
		for (int j = 0; j < n; j++) {
			if (i == 0 && j == 0) continue;
			x[i][j] = &x[0][0][i*n*o + j*o];
		}
	}
	return x;
}

/**
 * @brief Free a 3D array
 * 
 * @param array The 3D array to free
 */
void free_3d_array(double*** array) {
	free(array[0][0]);
	free(array[0]);
	free(array);
}

/**
 * @brief Free a 3D CUDA array
 *
 * @param array The 3D array to free
 */
void free_3d_cuda_array(double* array) {
    hipFree(array);
}

/**
 * @brief Allocate a 3D array that is addressable using square brackets - CUDA NOW
 *
 * @param m The first dimension of the array
 * @param n The second dimension of the array
 * @param o The third dimension of the array
 * @param array The arraay that needs to be allocated
 * @parm pitch can be read above
 */
void alloc_3d_cuda_array(int m, int n, int o, double **array, size_t *pitch) {
    hipMallocPitch((void **)array, pitch, n*o*sizeof(double), m);
    *pitch = (*pitch) / sizeof(double);
}

